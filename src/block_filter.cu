//
// Created by placek on 30.05.23.
//

#include <string>
#include <thrust/execution_policy.h>
#include <thrust/device_vector.h>

#include <config.hpp>

#include <block_filter.cuh>
#include <common.cuh>
#include "kmp.cuh"
#include "utils.cuh"

namespace filtering {

//
//    template<int Index>
//    size_t block_filter(std::string &lines) {
//        const auto length = lines.size() + 1;
//        const char *h_text = lines.c_str();
//        char *d_text;
//        bool *d_is_newline;
//        size_t *d_newline_positions;
//        bool *filter_result;
//
//        // malloc
//        CUDA_CHECK(hipMalloc(&d_text, length * sizeof(char)));
//        CUDA_CHECK(hipMalloc(&d_is_newline, length * sizeof(bool)));
//
//        // copy
//        CUDA_CHECK(hipMemcpy(d_text, h_text, length * sizeof(char), hipMemcpyHostToDevice));
//        CUDA_CHECK(hipMemset(d_is_newline, 0, length * sizeof(bool)));
//
//        // find newlines
//        auto block_size = (length - 1) / THREAD_SIZE + 1;
//        find_newlines_kernel<<<block_size, THREAD_SIZE>>>(d_text, length, d_is_newline);
//
//        CUDA_KERNEL_FINISH();
//
//        // count newlines
//        const auto num_newlines = thrust::reduce(thrust::device, d_is_newline, d_is_newline + length, 0);
//
//        CUDA_CHECK(hipMalloc(&d_newline_positions, num_newlines * sizeof(size_t)));
//
//        // find newlines positions
//        thrust::copy_if(thrust::device,
//                        thrust::make_counting_iterator<unsigned long>(0),
//                        thrust::make_counting_iterator(length),
//                        d_is_newline,
//                        d_newline_positions,
//                        thrust::identity<bool>());
//
//        CUDA_CHECK(hipMalloc(&filter_result, num_newlines * sizeof(bool)));
//        CUDA_CHECK(hipMemset(filter_result, 0, num_newlines * sizeof(bool)));
//
//        // find largest jsonl size
//        size_t *d_differences;
//        CUDA_CHECK(hipMalloc(&d_differences, num_newlines * sizeof(size_t)));
//
//        // set first difference as a value of first newline position
//        CUDA_CHECK(hipMemcpy(d_differences, d_newline_positions, sizeof(size_t), hipMemcpyDeviceToDevice));
//        thrust::transform(thrust::device,
//                          d_newline_positions + 1,
//                          d_newline_positions + num_newlines,
//                          d_newline_positions,
//                          d_differences + 1,
//                          thrust::minus<size_t>());
//        const auto max_jsonl_size = thrust::reduce(thrust::device, d_differences, d_differences + num_newlines, 0,
//                                                   thrust::maximum<size_t>());
//
//        // free
//        CUDA_CHECK(hipFree(d_differences));
//
//        // filter
//        block_size = num_newlines;
//        filter_block_per_json_kernel<Index><<<block_size, THREAD_SIZE, max_jsonl_size * sizeof(char) + 1>>>(d_text,
//                                                                                                        num_newlines,
//                                                                                                        d_newline_positions,
//                                                                                                        filter_result);
//
//        CUDA_KERNEL_FINISH();
//
//        // count filtered
//        const auto num_filtered = thrust::reduce(thrust::device, filter_result, filter_result + num_newlines, 0);
//
//        // free
//        CUDA_CHECK(hipFree(d_text));
//        CUDA_CHECK(hipFree(d_is_newline));
//        CUDA_CHECK(hipFree(d_newline_positions));
//        CUDA_CHECK(hipFree(filter_result));
//
//        return num_filtered;
//    }


}

//template size_t filtering::block_filter<configuration::Index>(std::string &lines);